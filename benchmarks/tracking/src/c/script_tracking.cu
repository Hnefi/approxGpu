/********************************
Author: Sravanthi Kota Venkata
 ********************************/

#include "tracking.h"
#include <iostream>
#include <fstream>
#include <string>

// LVA for interacting with pin
#ifdef APPROXIMATE
extern void LVA_FUNCTION(int type,void* start, void* end, int self) __attribute__ ((noinline));
extern void LVA_FUNCTION_RM(int type, void* start,void*end, int self) __attribute__ ((noinline));

extern void LVA_FUNCTION(int type, void* start, void* end, int self)
{ __asm__ __volatile__ ("xchg %dx,%dx"); }

extern void LVA_FUNCTION_RM(int type,void* start,void* end, int self)
{ __asm__ __volatile__ ("xchg %dx,%dx"); }
#endif

#define LVA_BX_INSTRUCTION __asm__ __volatile__ ("xchg %bx,%bx");

int main(int argc, char* argv[])
{
    int i, j, k, N_FEA, WINSZ, LK_ITER, rows, cols;
    int endR, endC;
    F2D *blurredImage, *previousFrameBlurred_level1, *previousFrameBlurred_level2, *blurred_level1, *blurred_level2;
    F2D *verticalEdgeImage, *horizontalEdgeImage, *verticalEdge_level1, *verticalEdge_level2, *horizontalEdge_level1, *horizontalEdge_level2, *interestPnt;
    F2D *lambda, *lambdaTemp, *features;
    I2D *Ic, *status;
    float SUPPRESION_RADIUS;
    F2D *newpoints;

    int numFind, m, n;
    F2D *np_temp;

    unsigned int* start, *end, *elapsed, *elt;
    char im1[100];
    int counter=2;
    float accuracy = 0.03;
    int count;

    if(argc < 3) 
    {
        printf("We need input image path AND training set file.\n");
        return -1;
    }
    std::string inputTexFile;
    sprintf(im1, "%s/bug_frames/1.bmp", argv[1]);
    char img1Name[100];
    sprintf(img1Name,"%s/bug_frames/1.bmp",argv[1]);
    inputTexFile.assign(argv[2]);

    N_FEA = 1600;
    WINSZ = 4;
    SUPPRESION_RADIUS = 10.0;
    LK_ITER = 20;

#ifdef test
    WINSZ = 2;
    N_FEA = 100;
    LK_ITER = 2;
    counter = 2;
    accuracy = 0.1;
#endif
#ifdef sim_fast
    WINSZ = 2;
    N_FEA = 100;
    LK_ITER = 2;
    counter = 4;
#endif
#ifdef sim
    WINSZ = 2;
    N_FEA = 200;
    LK_ITER = 2;
    counter = 4;
#endif
#ifdef sqcif
    WINSZ = 8;
    N_FEA = 500;
    LK_ITER = 15;
    counter = 2;
#endif
#ifdef qcif
    WINSZ = 12;
    N_FEA = 400;
    LK_ITER = 15;
    counter = 4;
#endif
#ifdef cif
    WINSZ = 20;
    N_FEA = 500;
    LK_ITER = 20;
    counter = 4;
#endif
#ifdef vga
    WINSZ = 32;
    N_FEA = 400;
    LK_ITER = 20;
    counter = 4;
#endif
#ifdef wuxga
    WINSZ = 64;
    N_FEA = 500;
    LK_ITER = 20;
    counter = 4;
#endif
#ifdef fullhd
    WINSZ = 48;
    N_FEA = 500;
    LK_ITER = 20;
    counter = 4;
#endif

    hipDeviceReset();
    printf("Input size\t\t- (%dx%d)\n", rows, cols);

    hipDeviceProp_t dev_props;
    // assume device 0
    HANDLE_ERROR( hipGetDeviceProperties(&dev_props,0) );

    // print some stuff
    printf("Current Device compute capability: %d.%d\n",dev_props.major,dev_props.minor);
    printf("1D texture memory limit (hipArray): %d\n",dev_props.maxTexture1D);

    // read in the texture training set from the input file
    float* big_arr = (float*)malloc( dev_props.maxTexture1D * sizeof(float)); 
    std::ifstream inputStream(inputTexFile.c_str());
    if ( !inputStream.is_open() ) return false;
    std::string raw_string;
    i = 0;
    while( !inputStream.eof() ) {
        getline(inputStream,raw_string);
        float value = atof(raw_string.c_str());
        //cout << "value: " << value << endl;
        big_arr[i] = value;
        i++;
    }
    printf("Total size of training input read from file: %d\n", i-1);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat); 
    hipArray* cuArray; 
    hipMallocArray(&cuArray, &channelDesc, i-1); 
    // Copy to device memory some data located at address h_data in host memory 
    hipMemcpyToArray(cuArray, 0, 0, big_arr,(i-1)*sizeof(float),hipMemcpyHostToDevice); 
    // Specify texture struct 
    hipResourceDesc resDesc; 
    memset(&resDesc, 0, sizeof(resDesc)); 
    resDesc.resType = hipResourceTypeArray; 
    resDesc.res.array.array = cuArray; 
    // Specify texture object parameters 
    struct hipTextureDesc texDesc; 
    memset(&texDesc, 0, sizeof(texDesc)); 
    texDesc.addressMode[0] = hipAddressModeMirror; 
    texDesc.filterMode = hipFilterModeLinear; 
    texDesc.readMode = hipReadModeElementType; 
    texDesc.normalizedCoords = 1; 
    // Create texture object 
    hipTextureObject_t texObj = 0; 
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);


    F2D* blurs[3];
    F2D* resizes[3];
    F2D* sobelx[3];
    F2D* sobely[3];

    // do this for all the rgb channels
    for(int arg = 2;arg >= 0;arg--) {
        Ic = readImage(im1,arg);
        rows = Ic->height;
        cols = Ic->width;
        ImagePyramid* preprocessed = createOutputImages(Ic,&texObj); // just need to define a struct to return 4 float* arrays
        //printf("After calling createImgPyramid...\n");
        //writeImgToFile(blurredImage,img1Name,"test.bmp");

        /** Scale down the image to build Image Pyramid. We find features across all scales of the image **/
        blurred_level1 = preprocessed->blurredImg;                   /** Scale 0 **/
        blurred_level2 = preprocessed->resizedImg;     /** Scale 1 **/
        horizontalEdgeImage = preprocessed->horizEdge;
        verticalEdgeImage = preprocessed->vertEdge;

        // copy the first image into the "saved pixels" to merge later
        blurs[arg] = fDeepCopy(blurred_level1);
        resizes[arg] = fDeepCopy(blurred_level2);
        sobelx[arg] = fDeepCopy(horizontalEdgeImage);
        sobely[arg] = fDeepCopy(verticalEdgeImage);
        destroyImgPyramid(preprocessed,0x0);
    }

    // write out the first image frame.
    writeImgToFile(blurs[2],blurs[1],blurs[0],img1Name,"blur_out.bmp");
    writeImgToFile(resizes[2],resizes[1],resizes[0],img1Name,"resize_out.bmp");
    writeImgToFile(sobelx[2],sobelx[1],sobelx[0],img1Name,"sobelx_out.bmp");
    writeImgToFile(sobely[2],sobely[1],sobely[0],img1Name,"sobely_out.bmp");

    /** Read input image **/
    Ic = readImage(im1);
    rows = Ic->height;
    cols = Ic->width;
    /* Other frames */
#define MAX_COUNTER     (4)
    I2D *Ics[MAX_COUNTER];
    ImagePyramid* newFramePyramids[MAX_COUNTER];
    hipStream_t frameStreams[MAX_COUNTER];

    /** Until now, we processed base frame. The following for loop processes other frames **/
    for(count=1; count<=counter; count++)
    {
        /** Read image **/
        sprintf(im1, "%s/bug_frames/%d.bmp", argv[1], count);
        Ics[count-1] = readImage(im1);
    }

    //start roi
    LVA_BX_INSTRUCTION;
    LVA_BX_INSTRUCTION;

    /** Start Timing **/
    start = photonStartTiming();

    /** IMAGE PRE-PROCESSING **/

    /** Blur the image to remove noise - weighted avergae filter **/

    ImagePyramid* preprocessed = createImgPyramid(Ic, 0,&texObj,false); // just need to define a struct to return 4 float* arrays
    //printf("After calling createImgPyramid...\n");

    blurredImage = preprocessed->blurredImg;
    //writeImgToFile(blurredImage,img1Name,"test.bmp");

    /** Scale down the image to build Image Pyramid. We find features across all scales of the image **/
    blurred_level1 = fDeepCopy(preprocessed->blurredImg);                   /** Scale 0 **/
    blurred_level2 = fDeepCopy(preprocessed->resizedImg);     /** Scale 1 **/
    horizontalEdgeImage = preprocessed->horizEdge;
    verticalEdgeImage = preprocessed->vertEdge;

    // copy the first image into the "saved pixels" to merge later

    /** Edge images are used for feature detection. So, using the verticalEdgeImage and horizontalEdgeImage images, we compute feature strength
      across all pixels. Lambda matrix is the feature strength matrix returned by calcGoodFeature **/

    lambda = calcGoodFeature(verticalEdgeImage, horizontalEdgeImage, verticalEdgeImage->width, verticalEdgeImage->height, WINSZ);
    endR = lambda->height;
    endC = lambda->width;
    lambdaTemp = fReshape(lambda, endR*endC, 1);

    /** We sort the lambda matrix based on the strengths **/
    /** Fill features matrix with top N_FEA features **/
    fFreeHandle(lambdaTemp);
    lambdaTemp = fillFeatures(lambda, N_FEA, WINSZ);
    features = fTranspose(lambdaTemp);

    /** Suppress features that have approximately similar strength and belong to close neighborhood **/
    interestPnt = getANMS(features, SUPPRESION_RADIUS);

    /** Refill interestPnt in features matrix **/
    fFreeHandle(features);
    features = fSetArray(2, interestPnt->height, 0);
    for(i=0; i<2; i++) {
        for(j=0; j<interestPnt->height; j++) {
            subsref(features,i,j) = subsref(interestPnt,j,i); 
        }
    } 
    /* commented out these frees to perform one big batch free on the returned image structure
       fFreeHandle(verticalEdgeImage);
       fFreeHandle(horizontalEdgeImage);
     */
    fFreeHandle(interestPnt);
    fFreeHandle(lambda);
    fFreeHandle(lambdaTemp);
    iFreeHandle(Ic);
    destroyImgPyramid(preprocessed,0x0);

    /** Until now, we processed base frame. The following for loop processes other frames **/
    for(count=1; count<=counter; count++)
    {
        newFramePyramids[count-1] = createImgPyramid(Ics[count-1],0,&texObj,false);

        Ic = Ics[count-1];
        rows = Ic->height;
        cols = Ic->width;

        //printf("Read image %d of dim %dx%d.\n",count,rows,cols);
        /* Start timing */
        //start = photonStartTiming();


        /** Blur image to remove noise **/
        blurredImage = newFramePyramids[count-1]->blurredImg;

        /** Blur image to remove noise **/
        previousFrameBlurred_level1 = fDeepCopy(blurred_level1);
        previousFrameBlurred_level2 = fDeepCopy(blurred_level2);

        //MARK - added these because i deep copied into previousFrame, and then can get rid of the old
        fFreeHandle(blurred_level1);
        fFreeHandle(blurred_level2);

        /** Image pyramid **/
        blurred_level1 = fDeepCopy(blurredImage);
        blurred_level2 = fDeepCopy(newFramePyramids[count-1]->resizedImg);

        verticalEdge_level1 = newFramePyramids[count-1]->vertEdge;
        verticalEdge_level2 = newFramePyramids[count-1]->vertEdge_small;
        horizontalEdge_level1 = newFramePyramids[count-1]->horizEdge;
        horizontalEdge_level2 = newFramePyramids[count-1]->horizEdge_small;

        newpoints = fSetArray(2, features->width, 0);

        /** Based on features computed in the previous frame, find correspondence in the current frame. "status" returns the index of corresponding features **/
        status = calcPyrLKTrack(previousFrameBlurred_level1, previousFrameBlurred_level2, verticalEdge_level1, verticalEdge_level2, horizontalEdge_level1, horizontalEdge_level2, blurred_level1, blurred_level2, features, features->width, WINSZ, accuracy, LK_ITER, newpoints);

        destroyImgPyramid(newFramePyramids[count-1], count);

        // left these ones (because they were just alloc'd in this loop
        fFreeHandle(previousFrameBlurred_level1);
        fFreeHandle(previousFrameBlurred_level2);

        /** Populate newpoints with features that had correspondence with previous frame features **/
        np_temp = fDeepCopy(newpoints);
        if(status->width > 0 )
        {
            k = 0;
            numFind=0;
            for(i=0; i<status->width; i++)
            {
                if( asubsref(status,i) == 1)
                    numFind++;
            }
            fFreeHandle(newpoints);
            newpoints = fSetArray(2, numFind, 0);

            for(i=0; i<status->width; i++)
            {
                if( asubsref(status,i) == 1)
                {
                    subsref(newpoints,0,k) = subsref(np_temp,0,i);
                    subsref(newpoints,1,k++) = subsref(np_temp,1,i);
                }
            }    
        }    

        iFreeHandle(status);
        iFreeHandle(Ic);
        fFreeHandle(np_temp);
        fFreeHandle(features);
        /** Populate newpoints into features **/
        features = fDeepCopy(newpoints);
        //printf("Printing features...\n");
        /*for(i = 0;i<features->height;i++) {
          for(j=0;j<features->width;j++) {
          printf("%f\t",subsref(features,i,j));
          }
          printf("\n");
          }*/

        fFreeHandle(newpoints);
    }
    /* Timing utils */
    end = photonEndTiming();
    elapsed = photonReportTiming(start, end);
    free(start);
    free(end);   

    //end roi
    LVA_BX_INSTRUCTION;


#ifdef CHECK   
    /* Self checking */
    {
        int ret=0;
        float tol = 2.0;
#ifdef GENERATE_OUTPUT
        fWriteMatrix(features, argv[1]);
#endif
        ret = fSelfCheck(features, argv[1], tol); 
        if (ret == -1)
            printf("Error in Tracking Map\n");
    }
#endif

    photonPrintTiming(elapsed);

    fFreeHandle(blurred_level1);
    fFreeHandle(blurred_level2);
    fFreeHandle(features);

    free(elapsed);
    free(big_arr);
    //free texture reference
    hipFreeArray(cuArray);
    hipDeviceReset();
    return 0;
}
