#include "hip/hip_runtime.h"
// Mark Sutherland, Josh San Miguel
//  - U of Toronto

// Global memory-based array image blur. non-optimized

#include "imageBlur_kernel_stage2.h"
#include "ghbFunctions.h"
#include <stdio.h>

#define RADIUS 2
#define SINGLEDIMINDEX(i,j,width) ((i)*(width) + (j))

__global__ void blurKernel_st2(float* outputPixels,float* intermediate, int* weightedKernel,uint width, uint height,hipTextureObject_t tref,int NUM_TEX /*, other arguments */)
{
    // assign id's
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int totalX = gridDim.x * blockDim.x;
    int totalY = gridDim.y * blockDim.y;

    /* Used to calculate the "ranges" each thread must span based on img size. */
    int xScale = width / totalX;
    if (width % totalX)
        xScale += 1;
    int yScale = height / totalY;
    if (height % totalY)
        yScale += 1;
    int xmod = width % totalX;
    int ymod = height % totalY;
    if (xScale > 1) { // each thread sweep more than 1 elem in x direction
       i *= xScale;
    }

    if (yScale > 1) { // same thing in y dimension
        j *= yScale;
    }
    float kernelSum = 16.0;
    extern __shared__ float ghb[]; // for per-thread local history
    int my_ghb_index = ((threadIdx.y * blockDim.x) + threadIdx.x) * 3;

    // still check for this in case of small img, not all threads need execute
    for (int idx = i; idx < (i + xScale); idx++) {
        if ( ((idx == i+xScale) && xmod == 0) ||
             ((idx == i+xScale) && width <= totalX )) break; // exact mult. corner case
        for (int jdx = j; jdx < (j + yScale); jdx++) { // over each element to proc
            if( ((jdx == j + yScale) && ymod == 0) ||
                ((jdx == j + yScale) && height <= totalY)) break; // same corner case
            float tmp = 0.0;

            if(idx < width-2 && idx > 1
               && jdx < height-2 && jdx > 1) { // bounds check #1
                  int curElement = SINGLEDIMINDEX(jdx,idx,width);
                  for(int ii = -RADIUS;ii <= (RADIUS - NUM_TEX);ii++) {
                      int location = curElement + (ii*width);
                      int filterWeightLoc = RADIUS + ii;
                      // bounds check #2 for surrounding pix
                      if (location < (width*height) && location >= 0) {
                          float loaded = intermediate[location];
                          tmp += loaded * weightedKernel[filterWeightLoc];
                          updateGHB(&(ghb[my_ghb_index]),loaded);
                      }
                  }

                  // finish up last few values with NUM_TEX reads
                  for(int ii = (RADIUS-NUM_TEX+1); ii <= RADIUS;ii++) {
                      int filterWeightLoc = RADIUS + ii;
                      float curValueHash = hashGHB(&ghb[my_ghb_index]);
                      float texVal = tex1D<float>(tref,curValueHash);
                      tmp += (ghb[my_ghb_index+2] + texVal) * weightedKernel[filterWeightLoc];
                  }
                float avg = tmp / kernelSum;
                outputPixels[curElement] = avg;
            }
        }
    }
}
