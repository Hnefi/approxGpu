#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include "sdvbs_common.h"

using std::cout;
using std::endl;

// sets up the global texture reference with allocation and release
bool createTextureReference(int rows, int cols, std::string inFile) {
#if 0

    // setup texture reference parameters
    tref.addressMode[0] = hipAddressModeClamp; 
    tref.filterMode = hipFilterModePoint;
    tref.normalized = true; // access with coordinates in range [0-1)

    // read in the texture training set from the input file
    float* big_arr = (float*)malloc( dev_props.maxTexture1D * sizeof(float)); 
    std::ifstream inputStream(inFile.c_str());
    if ( !inputStream.is_open() ) return false;
    std::string raw_string;
    int i = 0;
    while( !inputStream.eof() ) {
        getline(inputStream,raw_string);
        float value = atof(raw_string.c_str());
        //cout << "value: " << value << endl;
        big_arr[i] = value;
        i++;
    }
    printf("Total size of training input read from file: %d\n", i);

    // setup hipArray and place this into the device.
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
    //hipArray* my_arr; // my_arr is defined in ../kernels/texRef.h
    HANDLE_ERROR( hipMallocArray(&my_arr,&channelDesc,i) );

    // copy in
    HANDLE_ERROR( hipMemcpyToArray(my_arr,0,0,big_arr,i*sizeof(float),hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipBindTextureToArray(tref,my_arr,channelDesc) );

    float* copy_out = (float*)malloc( 100*sizeof(float));
    HANDLE_ERROR( hipMemcpyFromArray(copy_out,my_arr,0,0,3*sizeof(float),hipMemcpyDeviceToHost) );
    printf("%f, %f, %f\n",copy_out[0],copy_out[1],copy_out[2]);
    free(copy_out);
#endif
    return true;
}
