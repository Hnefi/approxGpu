#include "hip/hip_runtime.h"
// Mark Sutherland, Josh San Miguel
//  - Univ of Toronto

// Calls fast GPU implementations to create the requested GPU images (blur, resize, and sobel X/Y).

#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <fcntl.h>
#include <unistd.h>

#include "sdvbs_common.h"
#include "../kernels/imageBlur_kernel.h"
#include "../kernels/imageBlur_kernel_stage2.h"
#include "../kernels/imageResize_kernel.h"
#include "../kernels/imageResize_kernel_st2.h"
#include "../kernels/calcSobel_dX_kernel.h"
#include "../kernels/calcSobel_dY_kernel.h"

ImagePyramid* createImgPyramid(I2D* imageIn, hipStream_t d_stream,bool train_set = false)
{
    int rows, cols;
    rows = imageIn->height;
    cols = imageIn->width;
  
    // setup kernels, thread objects, and GPU memory
    int weightedKernel[5] = {1,4,6,4,1};
    int sobelKernel_1[3] = {1,2,1};
    int sobelKernel_2[3] = {1,0,-1};
    
    //dim3 nblocks(4,3);
    dim3 threadsPerBlock(32,32);

    // dynamically calculate how many thread blocks to launch
    int rowsIn = floor((rows+1)/8);
    int colsIn = floor((cols+1)/8);

    int resizedRows = floor((rows+1)/2);
    int resizedCols = floor((cols+1)/2);

    int nBlocksWide = colsIn/32;
    if (colsIn % 32) nBlocksWide++;
    int nBlocksTall = rowsIn/32;
    if (rowsIn % 32) nBlocksTall++;
    dim3 nblocks(nBlocksWide,nBlocksTall);
    //printf("Calculated block dimensions as: %d x %d\n",nBlocksWide,nBlocksTall);

    int* d_inputPixels;
    float* d_outputPixels;
    float* d_intermediate;
    int* d_weightedKernel,*sobel_kern_1,*sobel_kern_2;
    float* resizeInt, *dxInt, *dyInt, *dyInt_small, *dxInt_small;
    float* resizeOutput, *dxOutput, *dyOutput, *dxOutput_small, *dyOutput_small;

    float* threadReads, *threadHashes;
    float* reads, *hashes;
    int bytesForSmem = 32*32 * 3 * sizeof(float); // each thread gets 3 entries of 4 bytes each
    if(train_set == true) {
        reads = (float*) calloc(5*rows*cols,sizeof(float));
        hashes = (float*) calloc(5*rows*cols,sizeof(float));
        HANDLE_ERROR( hipMalloc((void**)&threadReads,5*rows*cols*sizeof(float)) );
        HANDLE_ERROR( hipMalloc((void**)&threadHashes,5*rows*cols*sizeof(float)) );
    }

    //Pin host memory array for greatest speed transfer.
    HANDLE_ERROR( hipHostRegister(&(imageIn->data[0]),rows*cols*sizeof(int),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&weightedKernel[0],5*sizeof(int),hipHostRegisterPortable) ) ;
    HANDLE_ERROR( hipHostRegister(&sobelKernel_1[0],3*sizeof(int),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&sobelKernel_2[0],3*sizeof(int),hipHostRegisterPortable) );

    // SET UP MEMORY - local data
    hipMalloc((void**)&(imageIn->d_weightedKernel),5*sizeof(int));
    hipMalloc((void**)&(imageIn->sobel_kern_1),3*sizeof(int));
    hipMalloc((void**)&(imageIn->sobel_kern_2),3*sizeof(int));
    d_weightedKernel = imageIn->d_weightedKernel;
    sobel_kern_1 = imageIn->sobel_kern_1;
    sobel_kern_2 = imageIn->sobel_kern_2;
    hipMemcpyAsync(d_weightedKernel,&(weightedKernel[0]),5*sizeof(int),hipMemcpyHostToDevice,d_stream);
    hipMemcpyAsync(sobel_kern_1,&(sobelKernel_1[0]),3*sizeof(int),hipMemcpyHostToDevice,d_stream);
    hipMemcpyAsync(sobel_kern_2,&(sobelKernel_2[0]),3*sizeof(int),hipMemcpyHostToDevice,d_stream);
    hipStreamSynchronize(d_stream);

    // SET UP MEMORY
    hipMalloc((void**)&(imageIn->d_inputPixels),rows*cols*sizeof(int));
    hipMalloc((void**)&(imageIn->d_outputPixels),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->d_intermediate),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->resizeInt),rows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dxInt),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyInt),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->resizeOutput),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dxOutput),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyOutput),rows*cols*sizeof(float));

    hipMalloc((void**)&(imageIn->dxOutput_small),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyOutput_small),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dxInt_small),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyInt_small),resizedRows*resizedCols*sizeof(float));
    
    d_inputPixels = imageIn->d_inputPixels;
    d_outputPixels = imageIn->d_outputPixels;
    d_intermediate = imageIn->d_intermediate;
    resizeInt = imageIn->resizeInt;
    dxInt = imageIn->dxInt;
    dyInt = imageIn->dyInt;
    dyInt_small = imageIn->dyInt_small;
    dxInt_small = imageIn->dxInt_small;
    resizeOutput = imageIn->resizeOutput;
    dxOutput = imageIn->dxOutput;
    dyOutput = imageIn->dyOutput;
    dxOutput_small = imageIn->dxOutput_small;
    dyOutput_small = imageIn->dyOutput_small;

    // Copy in input data and input kernels.
    hipMemcpyAsync(d_inputPixels,&(imageIn->data[0]),rows*cols*sizeof(int),hipMemcpyHostToDevice,d_stream);

    // clear outputs since we only access some of these pixels, others must be blank 
    hipMemsetAsync(d_outputPixels,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(d_intermediate,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(resizeOutput,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(resizeInt,0,rows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dxOutput,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dyOutput,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dxInt,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dyInt,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dxOutput_small,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dyOutput_small,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dxInt_small,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dyInt_small,0,resizedRows*resizedCols*sizeof(float),d_stream);

    /* Kernel call */
    blurKernel_st1<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(d_inputPixels,d_intermediate,d_weightedKernel,threadHashes,threadReads,cols,rows);
    blurKernel_st2<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(d_outputPixels,d_intermediate,d_weightedKernel,cols,rows);

    /* Call all kernels in one stream (order does not matter as they all read their input from d_outputPixels) */
    resizeKernel_st1<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(d_outputPixels,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols);
    resizeKernel_st2<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(resizeOutput,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols);

    /* Calc dX Sobel filter */
    calcSobel_dX_k1<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(d_outputPixels,dxInt,sobel_kern_1,sobel_kern_2,cols,rows);
    calcSobel_dX_k2<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(dxInt,dxOutput,sobel_kern_1,sobel_kern_2,cols,rows);

    calcSobel_dY_k1<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(d_outputPixels,dyInt,sobel_kern_1,sobel_kern_2,cols,rows);
    calcSobel_dY_k2<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(dyInt,dyOutput,sobel_kern_1,sobel_kern_2,cols,rows);

    /* Calc level 2 sobel filter (on resized images) */
    calcSobel_dX_k1<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(resizeOutput,dxInt_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);
    calcSobel_dX_k2<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(dxInt_small,dxOutput_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);

    calcSobel_dY_k1<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(resizeOutput,dyInt_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);
    calcSobel_dY_k2<<<nblocks,threadsPerBlock,bytesForSmem,d_stream>>>(dyInt_small,dyOutput_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);

    if(train_set == true) { 
        hipStreamSynchronize(d_stream); 
        // we are synched here, now we can print out the training set (if we are frame 0)
        HANDLE_ERROR( hipMemcpy(reads,threadReads,5*rows*cols*sizeof(float),hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy(hashes,threadHashes,5*rows*cols*sizeof(float),hipMemcpyDeviceToHost) );
        for(int i = 0;i < 5*rows*cols;i+=5) {
            printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i],reads[i]);
            printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i+1],reads[i+1]);
            printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i+2],reads[i+2]);
            printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i+3],reads[i+3]);
            printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i+4],reads[i+4]);
        }
        hipFree(threadHashes);
        hipFree(threadReads);
        free(reads);
        free(hashes);
    }

    // deep copy into the destination F2D structures
    ImagePyramid* retStruct = (ImagePyramid*)malloc(sizeof(ImagePyramid));
    // alloc these sub-arrays as pinned memory (required for copyAsync)
    retStruct->blurredImg = fSetArray(rows,cols,0);
    retStruct->resizedImg = fSetArray(resizedRows,resizedCols,0);
    retStruct->horizEdge = fSetArray(rows,cols,0);
    retStruct->vertEdge = fSetArray(rows,cols,0);
    retStruct->horizEdge_small = fSetArray(resizedRows,resizedCols,0);
    retStruct->vertEdge_small = fSetArray(resizedRows,resizedCols,0);
    retStruct->tmp = fSetArray(rows,cols,0);
    HANDLE_ERROR( hipHostRegister(&(retStruct->blurredImg->data[0]),rows*cols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->resizedImg->data[0]),resizedRows*resizedCols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->horizEdge->data[0]),rows*cols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->vertEdge->data[0]),rows*cols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->horizEdge_small->data[0]),resizedRows*resizedCols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->vertEdge_small->data[0]),resizedRows*resizedCols*sizeof(float),hipHostRegisterPortable) );

    hipMemcpyAsync((void*)&(retStruct->blurredImg->data[0]),d_outputPixels,rows*cols*sizeof(float),hipMemcpyDeviceToHost,d_stream);
    hipMemcpyAsync((void*)&(retStruct->resizedImg->data[0]),resizeOutput,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost,d_stream);
    hipMemcpyAsync((void*)&(retStruct->vertEdge->data[0]),dxOutput,rows*cols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   
    hipMemcpyAsync((void*)&(retStruct->horizEdge->data[0]),dyOutput,rows*cols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   
    hipMemcpyAsync((void*)&(retStruct->vertEdge_small->data[0]),dxOutput_small,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   
    hipMemcpyAsync((void*)&(retStruct->horizEdge_small->data[0]),dyOutput_small,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   

    // UNSET Host memory pinning - local data
    hipHostUnregister(&weightedKernel[0]);
    hipHostUnregister(&sobelKernel_1[0]);
    hipHostUnregister(&sobelKernel_2[0]);

    return retStruct;
}

void destroyImgPyramid(I2D* imageIn, ImagePyramid *retStruct)
{
    // UNSET Host memory pinning.
    hipHostUnregister(&(imageIn->data[0]));

    hipHostUnregister(&(retStruct->blurredImg->data[0]));
    hipHostUnregister(&(retStruct->resizedImg->data[0]));
    hipHostUnregister(&(retStruct->horizEdge->data[0]));
    hipHostUnregister(&(retStruct->vertEdge->data[0]));
    hipHostUnregister(&(retStruct->horizEdge_small->data[0]));
    hipHostUnregister(&(retStruct->vertEdge_small->data[0]));
    hipHostUnregister(&(retStruct->tmp->data[0]));

    hipFree(imageIn->d_weightedKernel);
    hipFree(imageIn->sobel_kern_1);
    hipFree(imageIn->sobel_kern_2);
    hipFree(imageIn->resizeInt);
    hipFree(imageIn->dxInt);
    hipFree(imageIn->dyInt);
    hipFree(imageIn->resizeOutput);
    hipFree(imageIn->dxOutput);
    hipFree(imageIn->dyOutput);
    hipFree(imageIn->d_inputPixels);
    hipFree(imageIn->d_outputPixels);
    hipFree(imageIn->d_intermediate);
    hipFree(imageIn->dxInt_small);
    hipFree(imageIn->dyInt_small);
    hipFree(imageIn->dxOutput_small);
    hipFree(imageIn->dyOutput_small);
}

