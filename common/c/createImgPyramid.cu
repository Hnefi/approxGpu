#include "hip/hip_runtime.h"
// Mark Sutherland, Josh San Miguel
//  - Univ of Toronto

// Calls fast GPU implementations to create the requested GPU images (blur, resize, and sobel X/Y).

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <ctype.h>
#include <fcntl.h>
#include <unistd.h>
#include <assert.h>

#include "sdvbs_common.h"
#include "../kernels/imageBlur_kernel.h"
#include "../kernels/imageBlur_kernel_stage2.h"
#include "../kernels/imageResize_kernel.h"
#include "../kernels/imageResize_kernel_st2.h"
#include "../kernels/calcSobel_dX_kernel.h"
#include "../kernels/calcSobel_dY_kernel.h"

using std::cout;
using std::endl;

ImagePyramid* createImgPyramid(I2D* imageIn, hipStream_t d_stream, hipTextureObject_t* texObj, bool train_set)
{
    int rows, cols;
    rows = imageIn->height;
    cols = imageIn->width;
    hipTextureObject_t objToKernel = *texObj;
  
    // setup kernels, thread objects, and GPU memory
    int weightedKernel[5] = {1,4,6,4,1};
    int sobelKernel_1[3] = {1,2,1};
    int sobelKernel_2[3] = {1,0,-1};
    
    //dim3 nblocks(4,3);
    dim3 threadsPerBlock(16,16);

    // dynamically calculate how many thread blocks to launch
    int rowsIn = floor((rows+1)/8);
    int colsIn = floor((cols+1)/8);

    int resizedRows = floor((rows+1)/2);
    int resizedCols = floor((cols+1)/2);

    int nBlocksWide = colsIn/16;
    if (colsIn % 16) nBlocksWide++;
    int nBlocksTall = rowsIn/16;
    if (rowsIn % 16) nBlocksTall++;
    dim3 nblocks(nBlocksWide,nBlocksTall);
    //printf("Calculated block dimensions as: %d x %d\n",nBlocksWide,nBlocksTall);

    int* d_inputPixels;
    float* d_outputPixels;
    float* d_origInput;
    float* d_intermediate;
    int* d_weightedKernel,*sobel_kern_1,*sobel_kern_2;
    float* resizeInt, *dxInt, *dyInt, *dyInt_small, *dxInt_small;
    float* resizeOutput, *dxOutput, *dyOutput, *dxOutput_small, *dyOutput_small;

    float* threadReads, *threadHashes;
    float* reads, *hashes;
    int bytesForSmem = 16*16 * 3 * sizeof(float); // each thread gets 3 entries of 4 bytes each
    if(train_set == true) {
        reads = (float*) calloc(3*rows*cols,sizeof(float));
        hashes = (float*) calloc(3*rows*cols,sizeof(float));
        HANDLE_ERROR( hipMalloc((void**)&threadReads,3*rows*cols*sizeof(float)) );
        HANDLE_ERROR( hipMalloc((void**)&threadHashes,3*rows*cols*sizeof(float)) );
        hipMemset(threadReads,0xff,3*rows*cols*sizeof(float));
        hipMemset(threadHashes,0xff,3*rows*cols*sizeof(float));
    }

    // SET UP MEMORY - local data
    hipMalloc((void**)&(imageIn->d_weightedKernel),5*sizeof(int));
    hipMalloc((void**)&(imageIn->sobel_kern_1),3*sizeof(int));
    hipMalloc((void**)&(imageIn->sobel_kern_2),3*sizeof(int));
    d_weightedKernel = imageIn->d_weightedKernel;
    sobel_kern_1 = imageIn->sobel_kern_1;
    sobel_kern_2 = imageIn->sobel_kern_2;
    hipMemcpy(d_weightedKernel,&(weightedKernel[0]),5*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(sobel_kern_1,&(sobelKernel_1[0]),3*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(sobel_kern_2,&(sobelKernel_2[0]),3*sizeof(int),hipMemcpyHostToDevice);

    // SET UP MEMORY
    hipMalloc((void**)&(imageIn->d_inputPixels),rows*cols*sizeof(int));
    hipMalloc((void**)&(imageIn->d_outputPixels),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->d_intermediate),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->resizeInt),rows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dxInt),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyInt),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->resizeOutput),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dxOutput),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyOutput),rows*cols*sizeof(float));

    hipMalloc((void**)&(imageIn->dxOutput_small),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyOutput_small),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dxInt_small),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyInt_small),resizedRows*resizedCols*sizeof(float));
    
    hipMalloc((void**)&d_origInput,rows*cols*sizeof(float));

    d_inputPixels = imageIn->d_inputPixels;
    d_outputPixels = imageIn->d_outputPixels;
    d_intermediate = imageIn->d_intermediate;
    resizeInt = imageIn->resizeInt;
    dxInt = imageIn->dxInt;
    dyInt = imageIn->dyInt;
    dyInt_small = imageIn->dyInt_small;
    dxInt_small = imageIn->dxInt_small;
    resizeOutput = imageIn->resizeOutput;
    dxOutput = imageIn->dxOutput;
    dyOutput = imageIn->dyOutput;
    dxOutput_small = imageIn->dxOutput_small;
    dyOutput_small = imageIn->dyOutput_small;


    //F2D* origPixelInput = fiDeepCopy(imageIn);

    // Copy in input data and input kernels.
    hipMemcpy(d_inputPixels,&(imageIn->data[0]),rows*cols*sizeof(int),hipMemcpyHostToDevice);
    //hipMemcpy(d_origInput,&(origPixelInput->data),rows*cols*sizeof(float),hipMemcpyHostToDevice);

    // clear outputs since we only access some of these pixels, others must be blank 
    hipMemset(d_outputPixels,0,rows*cols*sizeof(float));
    hipMemset(d_intermediate,0,rows*cols*sizeof(float));
    hipMemset(resizeOutput,0,resizedRows*resizedCols*sizeof(float));
    hipMemset(resizeInt,0,rows*resizedCols*sizeof(float));
    hipMemset(dxOutput,0,rows*cols*sizeof(float));
    hipMemset(dyOutput,0,rows*cols*sizeof(float));
    hipMemset(dxInt,0,rows*cols*sizeof(float));
    hipMemset(dyInt,0,rows*cols*sizeof(float));
    hipMemset(dxOutput_small,0,resizedRows*resizedCols*sizeof(float));
    hipMemset(dyOutput_small,0,resizedRows*resizedCols*sizeof(float));
    hipMemset(dxInt_small,0,resizedRows*resizedCols*sizeof(float));
    hipMemset(dyInt_small,0,resizedRows*resizedCols*sizeof(float));

    blurKernel_st1<<<nblocks,threadsPerBlock,bytesForSmem>>>(d_inputPixels,d_intermediate,d_weightedKernel,threadHashes,threadReads,cols,rows,objToKernel);
    blurKernel_st2<<<nblocks,threadsPerBlock,bytesForSmem>>>(d_outputPixels,d_intermediate,d_weightedKernel,cols,rows,objToKernel);

    resizeKernel_st1<<<nblocks,threadsPerBlock,bytesForSmem>>>(d_outputPixels,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols,objToKernel);
    resizeKernel_st2<<<nblocks,threadsPerBlock,bytesForSmem>>>(resizeOutput,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols,objToKernel);

    //TODO: this is outputting a training set.
    calcSobel_dX_k1<<<nblocks,threadsPerBlock,bytesForSmem>>>(d_origInput,dxInt,threadHashes,threadReads,sobel_kern_1,sobel_kern_2,cols,rows,objToKernel);
    calcSobel_dX_k2<<<nblocks,threadsPerBlock,bytesForSmem>>>(dxInt,dxOutput,sobel_kern_1,sobel_kern_2,cols,rows,objToKernel);

    calcSobel_dY_k1<<<nblocks,threadsPerBlock,bytesForSmem>>>(d_outputPixels,dyInt,sobel_kern_1,sobel_kern_2,cols,rows,objToKernel);
    calcSobel_dY_k2<<<nblocks,threadsPerBlock,bytesForSmem>>>(dyInt,dyOutput,sobel_kern_1,sobel_kern_2,cols,rows,objToKernel);

    calcSobel_dX_k1<<<nblocks,threadsPerBlock,bytesForSmem>>>(resizeOutput,dxInt_small,threadHashes,threadReads,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows,objToKernel);
    calcSobel_dX_k2<<<nblocks,threadsPerBlock,bytesForSmem>>>(dxInt_small,dxOutput_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows,objToKernel);

    calcSobel_dY_k1<<<nblocks,threadsPerBlock,bytesForSmem>>>(resizeOutput,dyInt_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows,objToKernel);
    calcSobel_dY_k2<<<nblocks,threadsPerBlock,bytesForSmem>>>(dyInt_small,dyOutput_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows,objToKernel);

    hipDeviceSynchronize();
    if(train_set == true) { 
        // we are synched here, now we can print out the training set (if we are frame 0)
        HANDLE_ERROR( hipMemcpy(reads,threadReads,3*rows*cols*sizeof(float),hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy(hashes,threadHashes,3*rows*cols*sizeof(float),hipMemcpyDeviceToHost) );
        for(int i = 0;i < 3*rows*cols;i+=3) {
            /*
            if(!(reads[i] != reads[i]))
                printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i],reads[i]);
            if(!(reads[i+1] != reads[i+1]))
                printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i+1],reads[i+1]);
                */
            if(!(reads[i+2] != reads[i+2]))
                printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i+2],reads[i+2]);
            /*
            printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i+3],reads[i+3]);
            printf("Global history hash [%0.5f], value: %0.5f\n",hashes[i+4],reads[i+4]);
            */
        }
        hipFree(threadHashes);
        hipFree(threadReads);
        free(reads);
        free(hashes);
    }

    // deep copy into the destination F2D structures
    //cout << "Creating image pyramid." << endl;
    ImagePyramid* retStruct = (ImagePyramid*)malloc(sizeof(ImagePyramid));
    // alloc these sub-arrays as pinned memory (required for copyAsync)
    retStruct->blurredImg = fSetArray(rows,cols,0);
    retStruct->resizedImg = fSetArray(resizedRows,resizedCols,0);
    retStruct->horizEdge = fSetArray(rows,cols,0);
    retStruct->vertEdge = fSetArray(rows,cols,0);
    retStruct->horizEdge_small = fSetArray(resizedRows,resizedCols,0);
    retStruct->vertEdge_small = fSetArray(resizedRows,resizedCols,0);

   
    hipMemcpy((void*)&(retStruct->blurredImg->data[0]),d_outputPixels,rows*cols*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy((void*)&(retStruct->resizedImg->data[0]),resizeOutput,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost);
    //printf("addr of vertEdge->data: %p\n",retStruct->vertEdge->data);
    //printf("addr of horizEdge->data: %p\n",retStruct->horizEdge->data);
    hipMemcpy((void*)retStruct->vertEdge->data,dxOutput,rows*cols*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy((void*)&(retStruct->horizEdge->data[0]),dyOutput,rows*cols*sizeof(float),hipMemcpyDeviceToHost);   
    hipMemcpy((void*)&(retStruct->vertEdge_small->data[0]),dxOutput_small,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost);   
    hipMemcpy((void*)&(retStruct->horizEdge_small->data[0]),dyOutput_small,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost);   

    hipFree(imageIn->d_weightedKernel);
    hipFree(imageIn->sobel_kern_1);
    hipFree(imageIn->sobel_kern_2);
    hipFree(imageIn->resizeInt);
    hipFree(imageIn->dxInt);
    hipFree(imageIn->dyInt);
    hipFree(imageIn->resizeOutput);
    hipFree(imageIn->dxOutput);
    hipFree(imageIn->dyOutput);
    hipFree(imageIn->d_inputPixels);
    hipFree(imageIn->d_outputPixels);
    hipFree(imageIn->d_intermediate);
    hipFree(imageIn->dxInt_small);
    hipFree(imageIn->dyInt_small);
    hipFree(imageIn->dxOutput_small);
    hipFree(imageIn->dyOutput_small);


    //hipFree(d_origInput);    
    //fFreeHandle(origPixelInput);
    
    return retStruct;
}

void destroyImgPyramid(ImagePyramid* retStruct, int imgNum )
{
    assert(retStruct != 0);
    //cout << "Destroying image pyramid for frame " << imgNum << endl;

    fFreeHandle(retStruct->blurredImg);
    fFreeHandle(retStruct->resizedImg);
    fFreeHandle(retStruct->horizEdge);
    fFreeHandle(retStruct->vertEdge);
    fFreeHandle(retStruct->horizEdge_small);
    fFreeHandle(retStruct->vertEdge_small);
    free(retStruct);
}

