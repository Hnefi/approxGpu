#include "hip/hip_runtime.h"
// Mark Sutherland, Josh San Miguel
//  - Univ of Toronto

// Calls fast GPU implementations to create the requested GPU images (blur, resize, and sobel X/Y).

#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <fcntl.h>
#include <unistd.h>

#include "sdvbs_common.h"
#include "../kernels/imageBlur_kernel.h"
#include "../kernels/imageBlur_kernel_stage2.h"
#include "../kernels/imageResize_kernel.h"
#include "../kernels/imageResize_kernel_st2.h"
//#include "../kernels/sobel_dX.h"
//#include "../kernels/sobel_dY.h"


ImagePyramid* createImgPyramid(I2D* imageIn)
{
    int rows, cols;
    rows = imageIn->height;
    cols = imageIn->width;
  
    // setup kernels, thread objects, and GPU memory
    int weightedKernel[5] = {1,4,6,4,1};
    int sobelKernel_1[3] = {1,0,-1};
    int sobelKernel_2[3] = {1,2,1};
    
    //dim3 nblocks(4,3);
    dim3 threadsPerBlock(32,32);

    // dynamically calculate how many thread blocks to launch
    int rowsIn = floor((rows+1)/4);
    int colsIn = floor((cols+1)/4);

    int resizedRows = floor((rows+1)/2);
    int resizedCols = floor((cols+1)/2);

    int nBlocksWide = colsIn/32;
    if (colsIn % 32) nBlocksWide++;
    int nBlocksTall = rowsIn/32;
    if (rowsIn % 32) nBlocksTall++;
    dim3 nblocks(nBlocksWide,nBlocksTall);
    //printf("Calculated block dimensions as: %d x %d\n",nBlocksWide,nBlocksTall);

    int* d_inputPixels;
    float* d_outputPixels;
    float* d_intermediate;
    int* d_weightedKernel;
    hipMalloc((void**)&d_inputPixels,rows*cols*sizeof(int));
    hipMalloc((void**)&d_outputPixels,rows*cols*sizeof(float));
    hipMalloc((void**)&d_intermediate,rows*cols*sizeof(float));
    hipMalloc((void**)&d_weightedKernel,5*sizeof(int));

    hipMemcpy(d_inputPixels,&(imageIn->data[0]),rows*cols*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_weightedKernel,&(weightedKernel[0]),5*sizeof(int),hipMemcpyHostToDevice);
    hipMemset(d_outputPixels,0,rows*cols*sizeof(float));
    hipMemset(d_intermediate,0,rows*cols*sizeof(float));

    // set up memory for other 3 images. first blur output serves as input here.
    float* resizeInt, *dxInt, *dyInt;
    float* resizeOutput, *dxOutput, *dyOutput;
    hipMalloc((void**)&resizeInt,rows*resizedCols*sizeof(float));
    hipMalloc((void**)&dxInt,rows*cols*sizeof(float));
    hipMalloc((void**)&dyInt,rows*cols*sizeof(float));
    hipMalloc((void**)&resizeOutput,resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&dxOutput,rows*cols*sizeof(float));
    hipMalloc((void**)&dyOutput,rows*cols*sizeof(float));

    // clear outputs since we only access some of these pixels, others are blank 
    hipMemset(resizeOutput,0,resizedRows*resizedCols*sizeof(float));
    hipMemset(resizeInt,0,rows*resizedCols*sizeof(float));
    hipMemset(dxOutput,0,rows*cols*sizeof(float));
    hipMemset(dyOutput,0,rows*cols*sizeof(float));

    /* Kernel call */
    blurKernel_st1<<<nblocks,threadsPerBlock>>>(d_inputPixels,d_intermediate,d_weightedKernel,cols,rows);
    blurKernel_st2<<<nblocks,threadsPerBlock>>>(d_outputPixels,d_intermediate,d_weightedKernel,cols,rows);

    /* Call all kernels in one stream (order does not matter as they all read their input from d_outputPixels) */
    resizeKernel_st1<<<nblocks,threadsPerBlock>>>(d_outputPixels,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols);
    resizeKernel_st2<<<nblocks,threadsPerBlock>>>(resizeOutput,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols);
    //sobelXFilter<<<nblocks,threadsPerBlock>>>(d_outputPixels,dxOutput,dxInt,sobelKernel_1,sobelKernel_2,rows,cols);
    //sobelYFilter<<<nblocks,threadsPerBlock>>>(d_outputPixels,dyOutput,dyInt,sobelKernel_2,sobelKernel_1,rows, cols);

    // synch back after this stream
    hipDeviceSynchronize();

    // deep copy into the destination F2D structures
    ImagePyramid* retStruct = (ImagePyramid*)malloc(sizeof(ImagePyramid));
    retStruct->blurredImg = fSetArray(rows,cols,0);
    retStruct->resizedImg = fSetArray(resizedRows,resizedCols,0);
    retStruct->horizEdge = fSetArray(rows,resizedCols,0);
    retStruct->vertEdge = fSetArray(rows,cols,0);
    hipMemcpy((void*)&(retStruct->blurredImg->data[0]),d_outputPixels,rows*cols*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy((void*)&(retStruct->resizedImg->data[0]),resizeOutput,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost);

    // TEMPORARY COPY FOR DEBUG.
    //hipMemcpy((void*)&(retStruct->horizEdge->data[0]),resizeInt,rows*resizedCols*sizeof(float),hipMemcpyDeviceToHost);
    //hipMemcpy((void*)&(retStruct->vertEdge->data[0]),d_intermediate,rows*cols*sizeof(float),hipMemcpyDeviceToHost);

    hipFree(resizeInt);
    hipFree(dxInt);
    hipFree(dyInt);
    hipFree(resizeOutput);
    hipFree(dxOutput);
    hipFree(dyOutput);
    hipFree(d_inputPixels);
    hipFree(d_outputPixels);
    hipFree(d_intermediate);
    hipFree(d_weightedKernel);

    return retStruct;
}

