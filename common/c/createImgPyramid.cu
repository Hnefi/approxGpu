#include "hip/hip_runtime.h"
// Mark Sutherland, Josh San Miguel
//  - Univ of Toronto

// Calls fast GPU implementations to create the requested GPU images (blur, resize, and sobel X/Y).

#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <fcntl.h>
#include <unistd.h>

#include "sdvbs_common.h"
#include "../kernels/imageBlur_kernel.h"
#include "../kernels/imageBlur_kernel_stage2.h"
#include "../kernels/imageResize_kernel.h"
#include "../kernels/imageResize_kernel_st2.h"
#include "../kernels/calcSobel_dX_kernel.h"
#include "../kernels/calcSobel_dY_kernel.h"

ImagePyramid* createImgPyramid(I2D* imageIn, hipStream_t d_stream)
{
    int rows, cols;
    rows = imageIn->height;
    cols = imageIn->width;
  
    // setup kernels, thread objects, and GPU memory
    int weightedKernel[5] = {1,4,6,4,1};
    int sobelKernel_1[3] = {1,2,1};
    int sobelKernel_2[3] = {1,0,-1};
    
    //dim3 nblocks(4,3);
    dim3 threadsPerBlock(32,32);

    // dynamically calculate how many thread blocks to launch
    int rowsIn = floor((rows+1)/4);
    int colsIn = floor((cols+1)/4);

    int resizedRows = floor((rows+1)/2);
    int resizedCols = floor((cols+1)/2);

    int nBlocksWide = colsIn/32;
    if (colsIn % 32) nBlocksWide++;
    int nBlocksTall = rowsIn/32;
    if (rowsIn % 32) nBlocksTall++;
    dim3 nblocks(nBlocksWide,nBlocksTall);
    //printf("Calculated block dimensions as: %d x %d\n",nBlocksWide,nBlocksTall);

    int* d_inputPixels;
    float* d_outputPixels;
    float* d_intermediate;
    int* d_weightedKernel,*sobel_kern_1,*sobel_kern_2;
    float* resizeInt, *dxInt, *dyInt, *dyInt_small, *dxInt_small;
    float* resizeOutput, *dxOutput, *dyOutput, *dxOutput_small, *dyOutput_small;

    //Pin host memory array for greatest speed transfer.
    HANDLE_ERROR( hipHostRegister(&(imageIn->data[0]),rows*cols*sizeof(int),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&weightedKernel[0],5*sizeof(int),hipHostRegisterPortable) ) ;
    HANDLE_ERROR( hipHostRegister(&sobelKernel_1[0],3*sizeof(int),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&sobelKernel_2[0],3*sizeof(int),hipHostRegisterPortable) );

    // SET UP MEMORY - local data
    hipMalloc((void**)&(imageIn->d_weightedKernel),5*sizeof(int));
    hipMalloc((void**)&(imageIn->sobel_kern_1),3*sizeof(int));
    hipMalloc((void**)&(imageIn->sobel_kern_2),3*sizeof(int));
    d_weightedKernel = imageIn->d_weightedKernel;
    sobel_kern_1 = imageIn->sobel_kern_1;
    sobel_kern_2 = imageIn->sobel_kern_2;
    hipMemcpyAsync(d_weightedKernel,&(weightedKernel[0]),5*sizeof(int),hipMemcpyHostToDevice,d_stream);
    hipMemcpyAsync(sobel_kern_1,&(sobelKernel_1[0]),3*sizeof(int),hipMemcpyHostToDevice,d_stream);
    hipMemcpyAsync(sobel_kern_2,&(sobelKernel_2[0]),3*sizeof(int),hipMemcpyHostToDevice,d_stream);
    hipStreamSynchronize(d_stream);

    // SET UP MEMORY
    hipMalloc((void**)&(imageIn->d_inputPixels),rows*cols*sizeof(int));
    hipMalloc((void**)&(imageIn->d_outputPixels),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->d_intermediate),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->resizeInt),rows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dxInt),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyInt),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->resizeOutput),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dxOutput),rows*cols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyOutput),rows*cols*sizeof(float));

    hipMalloc((void**)&(imageIn->dxOutput_small),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyOutput_small),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dxInt_small),resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&(imageIn->dyInt_small),resizedRows*resizedCols*sizeof(float));
    
    d_inputPixels = imageIn->d_inputPixels;
    d_outputPixels = imageIn->d_outputPixels;
    d_intermediate = imageIn->d_intermediate;
    resizeInt = imageIn->resizeInt;
    dxInt = imageIn->dxInt;
    dyInt = imageIn->dyInt;
    dyInt_small = imageIn->dyInt_small;
    dxInt_small = imageIn->dxInt_small;
    resizeOutput = imageIn->resizeOutput;
    dxOutput = imageIn->dxOutput;
    dyOutput = imageIn->dyOutput;
    dxOutput_small = imageIn->dxOutput_small;
    dyOutput_small = imageIn->dyOutput_small;

    // Copy in input data and input kernels.
    hipMemcpyAsync(d_inputPixels,&(imageIn->data[0]),rows*cols*sizeof(int),hipMemcpyHostToDevice,d_stream);

    // clear outputs since we only access some of these pixels, others must be blank 
    hipMemsetAsync(d_outputPixels,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(d_intermediate,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(resizeOutput,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(resizeInt,0,rows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dxOutput,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dyOutput,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dxInt,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dyInt,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dxOutput_small,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dyOutput_small,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dxInt_small,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dyInt_small,0,resizedRows*resizedCols*sizeof(float),d_stream);

    /* Kernel call */
    blurKernel_st1<<<nblocks,threadsPerBlock,0,d_stream>>>(d_inputPixels,d_intermediate,d_weightedKernel,cols,rows);
    blurKernel_st2<<<nblocks,threadsPerBlock,0,d_stream>>>(d_outputPixels,d_intermediate,d_weightedKernel,cols,rows);

    /* Call all kernels in one stream (order does not matter as they all read their input from d_outputPixels) */
    resizeKernel_st1<<<nblocks,threadsPerBlock,0,d_stream>>>(d_outputPixels,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols);
    resizeKernel_st2<<<nblocks,threadsPerBlock,0,d_stream>>>(resizeOutput,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols);

    /* Calc dX Sobel filter */
    calcSobel_dX_k1<<<nblocks,threadsPerBlock,0,d_stream>>>(d_outputPixels,dxInt,sobel_kern_1,sobel_kern_2,cols,rows);
    calcSobel_dX_k2<<<nblocks,threadsPerBlock,0,d_stream>>>(dxInt,dxOutput,sobel_kern_1,sobel_kern_2,cols,rows);

    calcSobel_dY_k1<<<nblocks,threadsPerBlock,0,d_stream>>>(d_outputPixels,dyInt,sobel_kern_1,sobel_kern_2,cols,rows);
    calcSobel_dY_k2<<<nblocks,threadsPerBlock,0,d_stream>>>(dyInt,dyOutput,sobel_kern_1,sobel_kern_2,cols,rows);

    /* Calc level 2 sobel filter (on resized images) */
    calcSobel_dX_k1<<<nblocks,threadsPerBlock,0,d_stream>>>(resizeOutput,dxInt_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);
    calcSobel_dX_k2<<<nblocks,threadsPerBlock,0,d_stream>>>(dxInt_small,dxOutput_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);

    calcSobel_dY_k1<<<nblocks,threadsPerBlock,0,d_stream>>>(resizeOutput,dyInt_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);
    calcSobel_dY_k2<<<nblocks,threadsPerBlock,0,d_stream>>>(dyInt_small,dyOutput_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);

    // deep copy into the destination F2D structures
    ImagePyramid* retStruct = (ImagePyramid*)malloc(sizeof(ImagePyramid));
    // alloc these sub-arrays as pinned memory (required for copyAsync)
    retStruct->blurredImg = fSetArray(rows,cols,0);
    retStruct->resizedImg = fSetArray(resizedRows,resizedCols,0);
    retStruct->horizEdge = fSetArray(rows,cols,0);
    retStruct->vertEdge = fSetArray(rows,cols,0);
    retStruct->horizEdge_small = fSetArray(resizedRows,resizedCols,0);
    retStruct->vertEdge_small = fSetArray(resizedRows,resizedCols,0);
    retStruct->tmp = fSetArray(rows,cols,0);
    HANDLE_ERROR( hipHostRegister(&(retStruct->blurredImg->data[0]),rows*cols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->resizedImg->data[0]),resizedRows*resizedCols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->horizEdge->data[0]),rows*cols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->vertEdge->data[0]),rows*cols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->horizEdge_small->data[0]),resizedRows*resizedCols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->vertEdge_small->data[0]),resizedRows*resizedCols*sizeof(float),hipHostRegisterPortable) );

    hipMemcpyAsync((void*)&(retStruct->blurredImg->data[0]),d_outputPixels,rows*cols*sizeof(float),hipMemcpyDeviceToHost,d_stream);
    hipMemcpyAsync((void*)&(retStruct->resizedImg->data[0]),resizeOutput,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost,d_stream);
    hipMemcpyAsync((void*)&(retStruct->vertEdge->data[0]),dxOutput,rows*cols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   
    hipMemcpyAsync((void*)&(retStruct->horizEdge->data[0]),dyOutput,rows*cols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   
    hipMemcpyAsync((void*)&(retStruct->vertEdge_small->data[0]),dxOutput_small,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   
    hipMemcpyAsync((void*)&(retStruct->horizEdge_small->data[0]),dyOutput_small,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   

    // UNSET Host memory pinning - local data
    hipHostUnregister(&weightedKernel[0]);
    hipHostUnregister(&sobelKernel_1[0]);
    hipHostUnregister(&sobelKernel_2[0]);

    return retStruct;
}

void destroyImgPyramid(I2D* imageIn, ImagePyramid *retStruct)
{
    // UNSET Host memory pinning.
    hipHostUnregister(&(imageIn->data[0]));

    hipHostUnregister(&(retStruct->blurredImg->data[0]));
    hipHostUnregister(&(retStruct->resizedImg->data[0]));
    hipHostUnregister(&(retStruct->horizEdge->data[0]));
    hipHostUnregister(&(retStruct->vertEdge->data[0]));
    hipHostUnregister(&(retStruct->horizEdge_small->data[0]));
    hipHostUnregister(&(retStruct->vertEdge_small->data[0]));
    hipHostUnregister(&(retStruct->tmp->data[0]));

    hipFree(imageIn->d_weightedKernel);
    hipFree(imageIn->sobel_kern_1);
    hipFree(imageIn->sobel_kern_2);
    hipFree(imageIn->resizeInt);
    hipFree(imageIn->dxInt);
    hipFree(imageIn->dyInt);
    hipFree(imageIn->resizeOutput);
    hipFree(imageIn->dxOutput);
    hipFree(imageIn->dyOutput);
    hipFree(imageIn->d_inputPixels);
    hipFree(imageIn->d_outputPixels);
    hipFree(imageIn->d_intermediate);
    hipFree(imageIn->dxInt_small);
    hipFree(imageIn->dyInt_small);
    hipFree(imageIn->dxOutput_small);
    hipFree(imageIn->dyOutput_small);
}

