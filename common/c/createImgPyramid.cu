#include "hip/hip_runtime.h"
// Mark Sutherland, Josh San Miguel
//  - Univ of Toronto

// Calls fast GPU implementations to create the requested GPU images (blur, resize, and sobel X/Y).

#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <fcntl.h>
#include <unistd.h>

#include "sdvbs_common.h"
#include "../kernels/imageBlur_kernel.h"
#include "../kernels/imageBlur_kernel_stage2.h"
#include "../kernels/imageResize_kernel.h"
#include "../kernels/imageResize_kernel_st2.h"
#include "../kernels/calcSobel_dX_kernel.h"
#include "../kernels/calcSobel_dY_kernel.h"

ImagePyramid* createImgPyramid(I2D* imageIn, hipStream_t d_stream)
{
    int rows, cols;
    rows = imageIn->height;
    cols = imageIn->width;
  
    // setup kernels, thread objects, and GPU memory
    int weightedKernel[5] = {1,4,6,4,1};
    int sobelKernel_1[3] = {1,2,1};
    int sobelKernel_2[3] = {1,0,-1};
    
    //dim3 nblocks(4,3);
    dim3 threadsPerBlock(32,32);

    // dynamically calculate how many thread blocks to launch
    int rowsIn = floor((rows+1)/4);
    int colsIn = floor((cols+1)/4);

    int resizedRows = floor((rows+1)/2);
    int resizedCols = floor((cols+1)/2);

    int nBlocksWide = colsIn/32;
    if (colsIn % 32) nBlocksWide++;
    int nBlocksTall = rowsIn/32;
    if (rowsIn % 32) nBlocksTall++;
    dim3 nblocks(nBlocksWide,nBlocksTall);
    //printf("Calculated block dimensions as: %d x %d\n",nBlocksWide,nBlocksTall);

    int* d_inputPixels;
    float* d_outputPixels;
    float* d_intermediate;
    int* d_weightedKernel,*sobel_kern_1,*sobel_kern_2;
    float* resizeInt, *dxInt, *dyInt, *dyInt_small, *dxInt_small;
    float* resizeOutput, *dxOutput, *dyOutput, *dxOutput_small, *dyOutput_small;

    //Pin host memory array for greatest speed transfer.
    HANDLE_ERROR( hipHostRegister(&(imageIn->data[0]),rows*cols*sizeof(int),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&weightedKernel[0],5*sizeof(int),hipHostRegisterPortable) ) ;
    HANDLE_ERROR( hipHostRegister(&sobelKernel_1[0],3*sizeof(int),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&sobelKernel_2[0],3*sizeof(int),hipHostRegisterPortable) );

    // SET UP MEMORY
    hipMalloc((void**)&d_inputPixels,rows*cols*sizeof(int));
    hipMalloc((void**)&d_outputPixels,rows*cols*sizeof(float));
    hipMalloc((void**)&d_intermediate,rows*cols*sizeof(float));
    hipMalloc((void**)&d_weightedKernel,5*sizeof(int));
    hipMalloc((void**)&resizeInt,rows*resizedCols*sizeof(float));
    hipMalloc((void**)&dxInt,rows*cols*sizeof(float));
    hipMalloc((void**)&dyInt,rows*cols*sizeof(float));
    hipMalloc((void**)&resizeOutput,resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&dxOutput,rows*cols*sizeof(float));
    hipMalloc((void**)&dyOutput,rows*cols*sizeof(float));
    hipMalloc((void**)&sobel_kern_1,3*sizeof(int));
    hipMalloc((void**)&sobel_kern_2,3*sizeof(int));

    hipMalloc((void**)&dxOutput_small,resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&dyOutput_small,resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&dxInt_small,resizedRows*resizedCols*sizeof(float));
    hipMalloc((void**)&dyInt_small,resizedRows*resizedCols*sizeof(float));

    // Copy in input data and input kernels.
    hipMemcpyAsync(d_inputPixels,&(imageIn->data[0]),rows*cols*sizeof(int),hipMemcpyHostToDevice,d_stream);
    // TODO: some way I don't have to copy in these tiny kernels
    hipMemcpyAsync(d_weightedKernel,&(weightedKernel[0]),5*sizeof(int),hipMemcpyHostToDevice,d_stream);
    hipMemcpyAsync(sobel_kern_1,&(sobelKernel_1[0]),3*sizeof(int),hipMemcpyHostToDevice,d_stream);
    hipMemcpyAsync(sobel_kern_2,&(sobelKernel_2[0]),3*sizeof(int),hipMemcpyHostToDevice,d_stream);

    // clear outputs since we only access some of these pixels, others must be blank 
    hipMemsetAsync(d_outputPixels,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(d_intermediate,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(resizeOutput,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(resizeInt,0,rows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dxOutput,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dyOutput,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dxInt,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dyInt,0,rows*cols*sizeof(float),d_stream);
    hipMemsetAsync(dxOutput_small,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dyOutput_small,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dxInt_small,0,resizedRows*resizedCols*sizeof(float),d_stream);
    hipMemsetAsync(dyInt_small,0,resizedRows*resizedCols*sizeof(float),d_stream);

    /* Kernel call */
    blurKernel_st1<<<nblocks,threadsPerBlock,0,d_stream>>>(d_inputPixels,d_intermediate,d_weightedKernel,cols,rows);
    blurKernel_st2<<<nblocks,threadsPerBlock,0,d_stream>>>(d_outputPixels,d_intermediate,d_weightedKernel,cols,rows);

    /* Call all kernels in one stream (order does not matter as they all read their input from d_outputPixels) */
    resizeKernel_st1<<<nblocks,threadsPerBlock,0,d_stream>>>(d_outputPixels,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols);
    resizeKernel_st2<<<nblocks,threadsPerBlock,0,d_stream>>>(resizeOutput,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols);

    /* Calc dX Sobel filter */
    calcSobel_dX_k1<<<nblocks,threadsPerBlock,0,d_stream>>>(d_outputPixels,dxInt,sobel_kern_1,sobel_kern_2,cols,rows);
    calcSobel_dX_k2<<<nblocks,threadsPerBlock,0,d_stream>>>(dxInt,dxOutput,sobel_kern_1,sobel_kern_2,cols,rows);

    calcSobel_dY_k1<<<nblocks,threadsPerBlock,0,d_stream>>>(d_outputPixels,dyInt,sobel_kern_1,sobel_kern_2,cols,rows);
    calcSobel_dY_k2<<<nblocks,threadsPerBlock,0,d_stream>>>(dyInt,dyOutput,sobel_kern_1,sobel_kern_2,cols,rows);

    /* Calc level 2 sobel filter (on resized images) */
    calcSobel_dX_k1<<<nblocks,threadsPerBlock,0,d_stream>>>(resizeOutput,dxInt_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);
    calcSobel_dX_k2<<<nblocks,threadsPerBlock,0,d_stream>>>(dxInt_small,dxOutput_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);

    calcSobel_dY_k1<<<nblocks,threadsPerBlock,0,d_stream>>>(resizeOutput,dyInt_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);
    calcSobel_dY_k2<<<nblocks,threadsPerBlock,0,d_stream>>>(dyInt_small,dyOutput_small,sobel_kern_1,sobel_kern_2,resizedCols,resizedRows);

    // deep copy into the destination F2D structures
    ImagePyramid* retStruct = (ImagePyramid*)malloc(sizeof(ImagePyramid));
    // alloc these sub-arrays as pinned memory (required for copyAsync)
    retStruct->blurredImg = fSetArray(rows,cols,0);
    retStruct->resizedImg = fSetArray(resizedRows,resizedCols,0);
    retStruct->horizEdge = fSetArray(rows,cols,0);
    retStruct->vertEdge = fSetArray(rows,cols,0);
    retStruct->horizEdge_small = fSetArray(resizedRows,resizedCols,0);
    retStruct->vertEdge_small = fSetArray(resizedRows,resizedCols,0);
    retStruct->tmp = fSetArray(rows,cols,0);
    HANDLE_ERROR( hipHostRegister(&(retStruct->blurredImg->data[0]),rows*cols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->resizedImg->data[0]),resizedRows*resizedCols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->horizEdge->data[0]),rows*cols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->vertEdge->data[0]),rows*cols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->horizEdge_small->data[0]),resizedRows*resizedCols*sizeof(float),hipHostRegisterPortable) );
    HANDLE_ERROR( hipHostRegister(&(retStruct->vertEdge_small->data[0]),resizedRows*resizedCols*sizeof(float),hipHostRegisterPortable) );

    hipMemcpyAsync((void*)&(retStruct->blurredImg->data[0]),d_outputPixels,rows*cols*sizeof(float),hipMemcpyDeviceToHost,d_stream);
    hipMemcpyAsync((void*)&(retStruct->resizedImg->data[0]),resizeOutput,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost,d_stream);
    hipMemcpyAsync((void*)&(retStruct->vertEdge->data[0]),dxOutput,rows*cols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   
    hipMemcpyAsync((void*)&(retStruct->horizEdge->data[0]),dyOutput,rows*cols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   
    hipMemcpyAsync((void*)&(retStruct->vertEdge_small->data[0]),dxOutput_small,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   
    hipMemcpyAsync((void*)&(retStruct->horizEdge_small->data[0]),dyOutput_small,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost,d_stream);   

    // TODO: DEFINITELY synchronize before freeing stuff.
    hipStreamSynchronize(d_stream);

    // UNSET Host memory pinning.
    hipHostUnregister(&(imageIn->data[0]));
    hipHostUnregister(&weightedKernel[0]);
    hipHostUnregister(&sobelKernel_1[0]);
    hipHostUnregister(&sobelKernel_2[0]);

    hipHostUnregister(&(retStruct->blurredImg->data[0]));
    hipHostUnregister(&(retStruct->resizedImg->data[0]));
    hipHostUnregister(&(retStruct->horizEdge->data[0]));
    hipHostUnregister(&(retStruct->vertEdge->data[0]));
    hipHostUnregister(&(retStruct->horizEdge_small->data[0]));
    hipHostUnregister(&(retStruct->vertEdge_small->data[0]));
    hipHostUnregister(&(retStruct->tmp->data[0]));

    hipFree(resizeInt);
    hipFree(dxInt);
    hipFree(dyInt);
    hipFree(resizeOutput);
    hipFree(dxOutput);
    hipFree(dyOutput);
    hipFree(d_inputPixels);
    hipFree(d_outputPixels);
    hipFree(d_intermediate);
    hipFree(d_weightedKernel);
    hipFree(sobel_kern_1);
    hipFree(sobel_kern_2);
    hipFree(dxInt_small);
    hipFree(dyInt_small);
    hipFree(dxOutput_small);
    hipFree(dyOutput_small);

    return retStruct;
}

