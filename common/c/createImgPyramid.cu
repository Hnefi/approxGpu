#include "hip/hip_runtime.h"
// Mark Sutherland, Josh San Miguel
//  - Univ of Toronto

// Calls fast GPU implementations to create the requested GPU images (blur, resize, and sobel X/Y).

#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <fcntl.h>
#include <unistd.h>

#include "sdvbs_common.h"
#include "../kernels/imageBlur_kernel.h"
#include "../kernels/imageResize_kernel.h"
//#include "../kernels/sobel_dX.h"
//#include "../kernels/sobel_dY.h"


ImagePyramid* createImgPyramid(I2D* imageIn)
{
    int rows, cols;
    rows = imageIn->height;
    cols = imageIn->width;
  
    // setup kernels, thread objects, and GPU memory
    int weightedKernel[5] = {1,4,6,4,1};
    int sobelKernel_1[3] = {1,0,-1};
    int sobelKernel_2[3] = {1,2,1};
    dim3 nblocks(4,3);
    dim3 threadsPerBlock(32,32);
    int* d_inputPixels;
    float* d_outputPixels;
    float* d_intermediate;
    int* d_weightedKernel;
    hipMalloc((void**)&d_inputPixels,rows*cols*sizeof(int));
    hipMalloc((void**)&d_outputPixels,rows*cols*sizeof(float));
    hipMalloc((void**)&d_intermediate,rows*cols*sizeof(float));
    hipMalloc((void**)&d_weightedKernel,5*sizeof(int));

    hipMemcpy(d_inputPixels,&(imageIn->data[0]),rows*cols*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_weightedKernel,&(weightedKernel[0]),5*sizeof(int),hipMemcpyHostToDevice);
    hipMemset(d_outputPixels,0,rows*cols*sizeof(float));

    /* Kernel call */
    weightedBlurKernel<<<nblocks,threadsPerBlock>>>(d_inputPixels,d_outputPixels,d_intermediate,d_weightedKernel,cols,rows);

    hipDeviceSynchronize();

    float* outputPixels = (float*)malloc(rows*cols*sizeof(float));
    hipMemcpy(outputPixels,d_outputPixels,rows*cols*sizeof(float),hipMemcpyDeviceToHost);

    // set up memory for other 3 images. first blur output serves as input here.
    float* resizeInt, *dxInt, *dyInt;
    float* resizeOutput, *dxOutput, *dyOutput;

    // resize is smaller than the blurred
    int resizedRows = floor((rows+1)/2);
    int resizedCols = floor((cols+1)/2);

    hipMalloc((void**)&resizeInt,rows*cols*sizeof(int));
    hipMalloc((void**)&dxInt,rows*cols*sizeof(int));
    hipMalloc((void**)&dyInt,rows*cols*sizeof(int));
    hipMalloc((void**)&resizeOutput,resizedRows*resizedCols*sizeof(int));
    hipMalloc((void**)&dxOutput,rows*cols*sizeof(int));
    hipMalloc((void**)&dyOutput,rows*cols*sizeof(int));

    // clear outputs since we only access some of these pixels, others are blank 
    hipMemset(resizeOutput,0,resizedRows*resizedCols*sizeof(float));
    hipMemset(dxOutput,0,rows*cols*sizeof(float));
    hipMemset(dyOutput,0,rows*cols*sizeof(float));

    /* Call all kernels in one stream (order does not matter as they all read their input from d_outputPixels) */
    imageResizeKernel<<<nblocks,threadsPerBlock>>>(d_outputPixels,resizeOutput,resizeInt,d_weightedKernel,rows,cols,resizedRows,resizedCols);
    //sobelXFilter<<<nblocks,threadsPerBlock>>>(d_outputPixels,dxOutput,dxInt,sobelKernel_1,sobelKernel_2,rows,cols);
    //sobelYFilter<<<nblocks,threadsPerBlock>>>(d_outputPixels,dyOutput,dyInt,sobelKernel_2,sobelKernel_1,rows, cols);

    // synch back after this stream
    hipDeviceSynchronize();

    // deep copy into the destination F2D structures
    ImagePyramid* retStruct = (ImagePyramid*)malloc(sizeof(ImagePyramid));
    retStruct->blurredImg = fSetArray(rows,cols,0);
    retStruct->resizedImg = fSetArray(resizedRows,resizedCols,0);
    retStruct->horizEdge = fSetArray(rows,cols,0);
    retStruct->vertEdge = fSetArray(rows,cols,0);
    memcpy((void*)&(retStruct->blurredImg->data[0]),(void*)&outputPixels[0],rows*cols*sizeof(float));
    hipMemcpy((void*)&(retStruct->resizedImg->data[0]),resizeOutput,resizedRows*resizedCols*sizeof(float),hipMemcpyDeviceToHost);

    free(outputPixels); // on the host size, we copied it into the retStruct so we didn't lose the image
    hipFree(resizeInt);
    hipFree(dxInt);
    hipFree(dyInt);
    hipFree(resizeOutput);
    hipFree(dxOutput);
    hipFree(dyOutput);
    hipFree(d_inputPixels);
    hipFree(d_outputPixels);
    hipFree(d_intermediate);
    hipFree(d_weightedKernel);

    return retStruct;
}

